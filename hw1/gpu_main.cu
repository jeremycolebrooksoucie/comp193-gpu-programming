#include "hip/hip_runtime.h"
/**************************************************************************
*
*           COMP 193
*           GPU programming 
*           Exercise 1 template 
*
**************************************************************************/

#include <hip/hip_runtime.h>
#include <hiprand.h>                 // includes random num stuff
#include <hiprand/hiprand_kernel.h>       	// more rand stuff
#include <hip/hip_texture_types.h>
#include "book.h"

#include <stdio.h>
#include "gpu_main.h"

/*************************************************************************/
void addGPU(int *a, int *b, int *c, unsigned long vecSize){

    printf("you can remove this print statement\n");
    // arrays to pass to gpu
    int *dev_a, *dev_b, *dev_c;
    HANDLE_ERROR( hipMalloc( (void **) &dev_a, vecSize * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void **) &dev_b, vecSize * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void **) &dev_c, vecSize * sizeof(int) ) );

    // copy into these arrays
    HANDLE_ERROR( hipMemcpy( dev_a, a, vecSize * sizeof(int), 
                               hipMemcpyHostToDevice));
    HANDLE_ERROR( hipMemcpy( dev_b, b, vecSize * sizeof(int), 
                                hipMemcpyHostToDevice));

    add<<<vecSize, 1>>>( dev_a, dev_b, dev_c, vecSize);

    // copy back to device to fill c with results
    HANDLE_ERROR( hipMemcpy( c, dev_c, vecSize * sizeof(int), 
                              hipMemcpyDeviceToHost) );
    // memory clean up
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );

  
}

/*************************************************************************/

/*
 * kernal function to add arrays in a and b
 */
__global__ void add(int *a, int *b, int *c, unsigned long vecSize)
{
    int tid = blockIdx.x;
    if (tid < vecSize) {
        c[tid] = a[tid] + b[tid];
    }
}
